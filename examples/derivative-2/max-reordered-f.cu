#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void curvi (double * __restrict__ in_r1, double *__restrict__ in_u1, double * __restrict__ in_u2, double *__restrict__ in_u3, double * __restrict__ in_mu, double * __restrict__ in_la, double * __restrict__ in_met1, double * __restrict__ in_met2, double * __restrict__ in_met3, double * __restrict__ in_met4, double * strx, double * stry, double c1, double c2, int N) {
	//Determing the block's indices
	int blockdim_k= (int)(blockDim.x);
	int k0 = (int)(blockIdx.x)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	double (*u1)[304][304] = (double (*)[304][304])in_u1;
	double (*u2)[304][304] = (double (*)[304][304])in_u2;
	double (*u3)[304][304] = (double (*)[304][304])in_u3;
	double (*mu)[304][304] = (double (*)[304][304])in_mu;
	double (*la)[304][304] = (double (*)[304][304])in_la;
	double (*r1)[304][304] = (double (*)[304][304])in_r1;
	double (*met1)[304][304] = (double (*)[304][304])in_met1;
	double (*met2)[304][304] = (double (*)[304][304])in_met2;
	double (*met3)[304][304] = (double (*)[304][304])in_met3;
	double (*met4)[304][304] = (double (*)[304][304])in_met4;

	if (j>=2 & k>=2 & j<=N-3 & k<=N-3) {
		for (int i=2; i<=N-3; i++) {
double _t_7_;
double _t_4_;
double _t_5_;
double _t_2_;
double _t_0_;
double _t_10_;
double _t_11_;
double _t_16_;
double _t_17_;
double _t_15_;
double _t_26_;
double _t_23_;
double _t_24_;
double _t_21_;
double _t_29_;
double _t_30_;
double _t_35_;
double _t_36_;
double _t_34_;
double _t_46_;
double _t_43_;
double _t_44_;
double _t_41_;
double _t_49_;
double _t_50_;
double _t_55_;
double _t_56_;
double _t_54_;
double _t_65_;
double _t_62_;
double _t_63_;
double _t_60_;
double _t_68_;
double _t_69_;
double _t_74_;
double _t_75_;
double _t_73_;
double r1ic0jc0kc0 = r1[i][j][k];
double _t_86_;
double _t_83_;
double _t_84_;
double _t_81_;
double _t_90_;
double _t_91_;
double _t_95_;
double _t_96_;
double _t_104_;
double _t_101_;
double _t_102_;
double _t_108_;
double _t_109_;
double _t_113_;
double _t_114_;
double _t_79_;
double _t_123_;
double _t_120_;
double _t_121_;
double _t_118_;
double _t_127_;
double _t_128_;
double _t_132_;
double _t_133_;
double _t_141_;
double _t_138_;
double _t_139_;
double _t_145_;
double _t_146_;
double _t_150_;
double _t_151_;
double _t_159_;
double _t_160_;
double _t_157_;
double _t_155_;
double _t_164_;
double _t_165_;
double _t_171_;
double _t_172_;
double _t_169_;
double _t_176_;
double _t_177_;
double _t_184_;
double _t_185_;
double _t_182_;
double _t_189_;
double _t_190_;
double _t_196_;
double _t_197_;
double _t_194_;
double _t_201_;
double _t_202_;

_t_7_ = 2.0 * mu[i][j][k+2];
_t_7_ += la[i][j][k+2];
_t_4_ = met1[i][j][k+2] * _t_7_ * met2[i][j][k+2];
_t_5_ = c2 * u1[i+2][j][k+2];
_t_5_ -= c2 * u1[i-2][j][k+2];
_t_5_ += c1 * u1[i+1][j][k+2];
_t_5_ -= c1 * u1[i-1][j][k+2];
_t_2_ = strx[i] * _t_4_ * _t_5_;
_t_0_ = c2 * _t_2_ * stry[j];
_t_10_ = met1[i][j][k+2] * mu[i][j][k+2] * met3[i][j][k+2];
_t_11_ = c2 * u2[i+2][j][k+2];
_t_11_ -= c2 * u2[i-2][j][k+2];
_t_11_ += c1 * u2[i+1][j][k+2];
_t_11_ -= c1 * u2[i-1][j][k+2];
_t_0_ += c2 * _t_10_ * _t_11_;
_t_16_ = met1[i][j][k+2] * mu[i][j][k+2] * met4[i][j][k+2];
_t_17_ = c2 * u3[i+2][j][k+2];
_t_17_ -= c2 * u3[i-2][j][k+2];
_t_17_ += c1 * u3[i+1][j][k+2];
_t_17_ -= c1 * u3[i-1][j][k+2];
_t_15_ = _t_16_ * _t_17_;
_t_0_ += c2 * _t_15_ * stry[j];
_t_26_ = 2.0 * mu[i][j][k-2];
_t_26_ += la[i][j][k-2];
_t_23_ = met1[i][j][k-2] * _t_26_ * met2[i][j][k-2];
_t_24_ = c2 * u1[i+2][j][k-2];
_t_24_ -= c2 * u1[i-2][j][k-2];
_t_24_ += c1 * u1[i+1][j][k-2];
_t_24_ -= c1 * u1[i-1][j][k-2];
_t_21_ = strx[i] * _t_23_ * _t_24_;
_t_0_ += c2 * _t_21_ * stry[j];
_t_29_ = met1[i][j][k-2] * mu[i][j][k-2] * met3[i][j][k-2];
_t_30_ = c2 * u2[i+2][j][k-2];
_t_30_ -= c2 * u2[i-2][j][k-2];
_t_30_ += c1 * u2[i+1][j][k-2];
_t_30_ -= c1 * u2[i-1][j][k-2];
_t_0_ += c2 * _t_29_ * _t_30_;
_t_35_ = met1[i][j][k-2] * mu[i][j][k-2] * met4[i][j][k-2];
_t_36_ = c2 * u3[i+2][j][k-2];
_t_36_ -= c2 * u3[i-2][j][k-2];
_t_36_ += c1 * u3[i+1][j][k-2];
_t_36_ -= c1 * u3[i-1][j][k-2];
_t_34_ = _t_35_ * _t_36_;
_t_0_ += c2 * _t_34_ * stry[j];
_t_46_ = 2.0 * mu[i][j][k+1];
_t_46_ += la[i][j][k+1];
_t_43_ = met1[i][j][k+1] * _t_46_ * met2[i][j][k+1];
_t_44_ = c2 * u1[i+2][j][k+1];
_t_44_ -= c2 * u1[i-2][j][k+1];
_t_44_ += c1 * u1[i+1][j][k+1];
_t_44_ -= c1 * u1[i-1][j][k+1];
_t_41_ = strx[i+2] * _t_43_ * _t_44_;
_t_0_ += c1 * _t_41_ * stry[j];
_t_49_ = met1[i][j][k+1] * mu[i][j][k+1] * met3[i][j][k+1];
_t_50_ = c2 * u2[i+2][j][k+1];
_t_50_ -= c2 * u2[i-2][j][k+1];
_t_50_ += c1 * u2[i+1][j][k+1];
_t_50_ -= c1 * u2[i-1][j][k+1];
_t_0_ += c1 * _t_49_ * _t_50_;
_t_55_ = met1[i][j][k+1] * mu[i][j][k+1] * met4[i][j][k+1];
_t_56_ = c2 * u3[i+2][j][k+1];
_t_56_ -= c2 * u3[i-2][j][k+1];
_t_56_ += c1 * u3[i+1][j][k+1];
_t_56_ -= c1 * u3[i-1][j][k+1];
_t_54_ = _t_55_ * _t_56_;
_t_0_ += c1 * _t_54_ * stry[j];
_t_65_ = 2.0 * mu[i][j][k-1];
_t_65_ += la[i][j][k-1];
_t_62_ = met1[i][j][k-1] * _t_65_ * met2[i][j][k-1];
_t_63_ = c2 * u1[i+2][j][k-1];
_t_63_ -= c2 * u1[i-2][j][k-1];
_t_63_ += c1 * u1[i+1][j][k-1];
_t_63_ -= c1 * u1[i-1][j][k-1];
_t_60_ = strx[i-2] * _t_62_ * _t_63_;
_t_0_ += c1 * _t_60_ * stry[j];
_t_68_ = met1[i][j][k-1] * mu[i][j][k-1] * met3[i][j][k-1];
_t_69_ = c2 * u2[i+2][j][k-1];
_t_69_ -= c2 * u2[i-2][j][k-1];
_t_69_ += c1 * u2[i+1][j][k-1];
_t_69_ -= c1 * u2[i-1][j][k-1];
_t_0_ += c1 * _t_68_ * _t_69_;
_t_74_ = met1[i][j][k-1] * mu[i][j][k-1] * met4[i][j][k-1];
_t_75_ = c2 * u3[i+2][j][k-1];
_t_75_ -= c2 * u3[i-2][j][k-1];
_t_75_ += c1 * u3[i+1][j][k-1];
_t_75_ -= c1 * u3[i-1][j][k-1];
_t_73_ = _t_74_ * _t_75_;
_t_0_ += c1 * _t_73_ * stry[j];
r1ic0jc0kc0 += _t_0_;
_t_86_ = 2.0 * mu[i+2][j][k];
_t_86_ += la[i+2][j][k];
_t_83_ = met1[i+2][j][k] * _t_86_ * met2[i+2][j][k];
_t_84_ = c2 * u1[i+2][j][k+2];
_t_84_ -= c2 * u1[i+2][j][k-2];
_t_84_ += c1 * u1[i+2][j][k+1];
_t_84_ -= c1 * u1[i+2][j][k-1];
_t_81_ = strx[i] * _t_83_ * _t_84_;
_t_90_ = met1[i+2][j][k] * la[i+2][j][k] * met3[i+2][j][k];
_t_91_ = c2 * u2[i+2][j][k+2];
_t_91_ -= c2 * u2[i+2][j][k-2];
_t_91_ += c1 * u2[i+2][j][k+1];
_t_91_ -= c1 * u2[i+2][j][k-1];
_t_81_ += stry[j] * _t_90_ * _t_91_;
_t_95_ = met1[i+2][j][k] * la[i+2][j][k] * met4[i+2][j][k];
_t_96_ = c2 * u3[i+2][j][k+2];
_t_96_ -= c2 * u3[i+2][j][k-2];
_t_96_ += c1 * u3[i+2][j][k+1];
_t_96_ -= c1 * u3[i+2][j][k-1];
_t_81_ += _t_95_ * _t_96_;
_t_104_ = 2.0 * mu[i-2][j][k];
_t_104_ += la[i-2][j][k];
_t_101_ = met1[i-2][j][k] * _t_104_ * met2[i-2][j][k];
_t_102_ = c2 * u1[i-2][j][k+2];
_t_102_ -= c2 * u1[i-2][j][k-2];
_t_102_ += c1 * u1[i-2][j][k+1];
_t_102_ -= c1 * u1[i-2][j][k-1];
_t_81_ += strx[i] * _t_101_ * _t_102_;
_t_108_ = met1[i-2][j][k] * la[i-2][j][k] * met3[i-2][j][k];
_t_109_ = c2 * u2[i-2][j][k+2];
_t_109_ -= c2 * u2[i-2][j][k-2];
_t_109_ += c1 * u2[i-2][j][k+1];
_t_109_ -= c1 * u2[i-2][j][k-1];
_t_81_ += stry[j] * _t_108_ * _t_109_;
_t_113_ = met1[i-2][j][k] * la[i-2][j][k] * met4[i-2][j][k];
_t_114_ = c2 * u3[i-2][j][k+2];
_t_114_ -= c2 * u3[i-2][j][k-2];
_t_114_ += c1 * u3[i-2][j][k+1];
_t_114_ -= c1 * u3[i-2][j][k-1];
_t_81_ += _t_113_ * _t_114_;
_t_79_ = stry[j] * c2 * _t_81_;
_t_123_ = 2.0 * mu[i+1][j][k];
_t_123_ += la[i+1][j][k];
_t_120_ = met1[i+1][j][k] * _t_123_ * met2[i+1][j][k];
_t_121_ = c2 * u1[i+1][j][k+2];
_t_121_ -= c2 * u1[i+1][j][k-2];
_t_121_ += c1 * u1[i+1][j][k+1];
_t_121_ -= c1 * u1[i+1][j][k-1];
_t_118_ = strx[i] * _t_120_ * _t_121_;
_t_127_ = met1[i+1][j][k] * la[i+1][j][k] * met3[i+1][j][k];
_t_128_ = c2 * u2[i+1][j][k+2];
_t_128_ -= c2 * u2[i+1][j][k-2];
_t_128_ += c1 * u2[i+1][j][k+1];
_t_128_ -= c1 * u2[i+1][j][k-1];
_t_118_ += stry[j] * _t_127_ * _t_128_;
_t_132_ = met1[i+1][j][k] * la[i+1][j][k] * met4[i+1][j][k];
_t_133_ = c2 * u3[i+1][j][k+2];
_t_133_ -= c2 * u3[i+1][j][k-2];
_t_133_ += c1 * u3[i+1][j][k+1];
_t_133_ -= c1 * u3[i+1][j][k-1];
_t_118_ += _t_132_ * _t_133_;
_t_141_ = 2.0 * mu[i-1][j][k];
_t_141_ += la[i-1][j][k];
_t_138_ = met1[i-1][j][k] * _t_141_ * met2[i-1][j][k];
_t_139_ = c2 * u1[i-1][j][k+2];
_t_139_ -= c2 * u1[i-1][j][k-2];
_t_139_ += c1 * u1[i-1][j][k+1];
_t_139_ -= c1 * u1[i-1][j][k-1];
_t_118_ += strx[i] * _t_138_ * _t_139_;
_t_145_ = met1[i-1][j][k] * la[i-1][j][k] * met3[i-1][j][k];
_t_146_ = c2 * u2[i-1][j][k+2];
_t_146_ -= c2 * u2[i-1][j][k-2];
_t_146_ += c1 * u2[i-1][j][k+1];
_t_146_ -= c1 * u2[i-1][j][k-1];
_t_118_ += stry[j] * _t_145_ * _t_146_;
_t_150_ = met1[i-1][j][k] * la[i-1][j][k] * met4[i-1][j][k];
_t_151_ = c2 * u3[i-1][j][k+2];
_t_151_ -= c2 * u3[i-1][j][k-2];
_t_151_ += c1 * u3[i-1][j][k+1];
_t_151_ -= c1 * u3[i-1][j][k-1];
_t_118_ += _t_150_ * _t_151_;
_t_79_ += stry[j] * c1 * _t_118_;
r1ic0jc0kc0 += _t_79_;
_t_159_ = met1[i][j][k+2] * mu[i][j][k+2] * met3[i][j][k+2];
_t_160_ = c2 * u1[i][j+2][k+2];
_t_160_ -= c2 * u1[i][j-2][k+2];
_t_160_ += c1 * u1[i][j+1][k+2];
_t_160_ -= c1 * u1[i][j-1][k+2];
_t_157_ = stry[j+2] * _t_159_ * _t_160_;
_t_155_ = c2 * _t_157_ * strx[i];
_t_164_ = met1[i][j][k+2] * la[i][j][k+2] * met2[i][j][k+2];
_t_165_ = c2 * u2[i][j+2][k+2];
_t_165_ -= c2 * u2[i][j-2][k+2];
_t_165_ += c1 * u2[i][j+1][k+2];
_t_165_ -= c1 * u2[i][j-1][k+2];
_t_155_ += c2 * _t_164_ * _t_165_;
_t_171_ = met1[i][j][k-2] * mu[i][j][k-2] * met3[i][j][k-2];
_t_172_ = c2 * u1[i][j+2][k-2];
_t_172_ -= c2 * u1[i][j-2][k-2];
_t_172_ += c1 * u1[i][j+1][k-2];
_t_172_ -= c1 * u1[i][j-1][k-2];
_t_169_ = stry[j] * _t_171_ * _t_172_;
_t_155_ += c2 * _t_169_ * strx[i];
_t_176_ = met1[i][j][k-2] * la[i][j][k-2] * met2[i][j][k-2];
_t_177_ = c2 * u2[i][j+2][k-2];
_t_177_ -= c2 * u2[i][j-2][k-2];
_t_177_ += c1 * u2[i][j+1][k-2];
_t_177_ -= c1 * u2[i][j-1][k-2];
_t_155_ += c2 * _t_176_ * _t_177_;
_t_184_ = met1[i][j][k+1] * mu[i][j][k+1] * met3[i][j][k+1];
_t_185_ = c2 * u1[i][j+2][k+1];
_t_185_ -= c2 * u1[i][j-2][k+1];
_t_185_ += c1 * u1[i][j+1][k+1];
_t_185_ -= c1 * u1[i][j-1][k+1];
_t_182_ = stry[j-2] * _t_184_ * _t_185_;
_t_155_ += c1 * _t_182_ * strx[i];
_t_189_ = met1[i][j][k+1] * la[i][j][k+1] * met2[i][j][k+1];
_t_190_ = c2 * u2[i][j+2][k+1];
_t_190_ -= c2 * u2[i][j-2][k+1];
_t_190_ += c1 * u2[i][j+1][k+1];
_t_190_ -= c1 * u2[i][j-1][k+1];
_t_155_ += c1 * _t_189_ * _t_190_;
_t_196_ = met1[i][j][k-1] * mu[i][j][k-1] * met3[i][j][k-1];
_t_197_ = c2 * u1[i][j+2][k-1];
_t_197_ -= c2 * u1[i][j-2][k-1];
_t_197_ += c1 * u1[i][j+1][k-1];
_t_197_ -= c1 * u1[i][j-1][k-1];
_t_194_ = stry[j] * _t_196_ * _t_197_;
_t_155_ += c1 * _t_194_ * strx[i];
_t_201_ = met1[i][j][k-1] * la[i][j][k-1] * met2[i][j][k-1];
_t_202_ = c2 * u2[i][j+2][k-1];
_t_202_ -= c2 * u2[i][j-2][k-1];
_t_202_ += c1 * u2[i][j+1][k-1];
_t_202_ -= c1 * u2[i][j-1][k-1];
_t_155_ += c1 * _t_201_ * _t_202_;
r1ic0jc0kc0 += _t_155_;
r1[i][j][k] = r1ic0jc0kc0;

			r1[i][j][k] += c2*(
					mu[i][j+2][k]*met3[i][j+2][k]*met1[i][j+2][k]*(
						c2*(u1[i][j+2][k+2]-u1[i][j+2][k-2]) +
						c1*(u1[i][j+2][k+1]-u1[i][j+2][k-1])   )*stry[j+1]*strx[i]
					+ mu[i][j+2][k]*met2[i][j+2][k]*met1[i][j+2][k]*(
						c2*(u2[i][j+2][k+2]-u2[i][j+2][k-2]) +
						c1*(u2[i][j+2][k+1]-u2[i][j+2][k-1])  )
					+ ( mu[i][j-2][k]*met3[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u1[i][j-2][k+2]-u1[i][j-2][k-2]) +
							c1*(u1[i][j-2][k+1]-u1[i][j-2][k-1])  )*stry[j]*strx[i]
						+ mu[i][j-2][k]*met2[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u2[i][j-2][k+2]-u2[i][j-2][k-2]) +
							c1*(u2[i][j-2][k+1]-u2[i][j-2][k-1])   ) )
					) + c1*(
						mu[i][j+1][k]*met3[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u1[i][j+1][k+2]-u1[i][j+1][k-2]) +
							c1*(u1[i][j+1][k+1]-u1[i][j+1][k-1]) )*stry[j-1]*strx[i]
						+ mu[i][j+1][k]*met2[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u2[i][j+1][k+2]-u2[i][j+1][k-2]) +
							c1*(u2[i][j+1][k+1]-u2[i][j+1][k-1]) )
						+ ( mu[i][j-1][k]*met3[i][j-1][k]*met1[i][j-1][k]*(
								c2*(u1[i][j-1][k+2]-u1[i][j-1][k-2]) +
								c1*(u1[i][j-1][k+1]-u1[i][j-1][k-1]) )*stry[j]*strx[i]
							+ mu[i][j-1][k]*met2[i][j-1][k]*met1[i][j-1][k]*(
								c2*(u2[i][j-1][k+2]-u2[i][j-1][k-2]) +
								c1*(u2[i][j-1][k+1]-u2[i][j-1][k-1]) ) ) );

			r1[i][j][k] +=
				c2*(  mu[i][j+2][k]*met1[i][j+2][k]*met1[i][j+2][k]*(
							c2*(u2[i+2][j+2][k]-u2[i-2][j+2][k]) +
							c1*(u2[i+1][j+2][k]-u2[i-1][j+2][k])    )
						+  mu[i][j-2][k]*met1[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u2[i+2][j-2][k]-u2[i-2][j-2][k])+
							c1*(u2[i+1][j-2][k]-u2[i-1][j-2][k])     )
				   ) +
				c1*(  mu[i][j+1][k]*met1[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u2[i+2][j+1][k]-u2[i-2][j+1][k]) +
							c1*(u2[i+1][j+1][k]-u2[i-1][j+1][k])  )
						+ mu[i][j-1][k]*met1[i][j-1][k]*met1[i][j-1][k]*(
							c2*(u2[i+2][j-1][k]-u2[i-2][j-1][k]) +
							c1*(u2[i+1][j-1][k]-u2[i-1][j-1][k])))
				+
				c2*(  la[i+2][j][k]*met1[i+2][j][k]*met1[i+2][j][k]*(
							c2*(u2[i+2][j+2][k]-u2[i+2][j-2][k]) +
							c1*(u2[i+2][j+1][k]-u2[i+2][j-1][k])    )
						+ la[i-2][j][k]*met1[i-2][j][k]*met1[i-2][j][k]*(
							c2*(u2[i-2][j+2][k]-u2[i-2][j-2][k])+
							c1*(u2[i-2][j+1][k]-u2[i-2][j-1][k])     )
				   ) +
				c1*(  la[i+1][j][k]*met1[i+1][j][k]*met1[i+1][j][k]*(
							c2*(u2[i+1][j+2][k]-u2[i+1][j-2][k]) +
							c1*(u2[i+1][j+1][k]-u2[i+1][j-1][k])  )
						+ la[i-1][j][k]*met1[i-1][j][k]*met1[i-1][j][k]*(
							c2*(u2[i-1][j+2][k]-u2[i-1][j-2][k]) +
							c1*(u2[i-1][j+1][k]-u2[i-1][j-1][k])));

		} 
	}
}

extern "C" void host_code (double *h_r1, double *h_u1, double *h_u2, double *h_u3,  double *h_mu, double *h_la, double *h_met1, double *h_met2, double *h_met3, double *h_met4, double *h_strx, double *h_stry, double c1, double c2, int N) {
	double *r1;
	hipMalloc (&r1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for r1\n");
	hipMemcpy (r1, h_r1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u1;
	hipMalloc (&u1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u1\n");
	hipMemcpy (u1, h_u1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u2;
	hipMalloc (&u2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u2\n");
	hipMemcpy (u2, h_u2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u3;
	hipMalloc (&u3, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u3\n");
	hipMemcpy (u3, h_u3, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *mu;
	hipMalloc (&mu, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *la;
	hipMalloc (&la, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met1;
	hipMalloc (&met1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met1\n");
	hipMemcpy (met1, h_met1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met2;
	hipMalloc (&met2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met2\n");
	hipMemcpy (met2, h_met2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met3;
	hipMalloc (&met3, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met3\n");
	hipMemcpy (met3, h_met3, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met4;
	hipMalloc (&met4, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met4\n");
	hipMemcpy (met4, h_met4, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double)*N, hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), 1);

	curvi <<<gridconfig, blockconfig>>> (r1, u1, u2, u3, mu, la, met1, met2, met3, met4, strx, stry, c1, c2, N);
	hipMemcpy (h_r1, r1, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
}
