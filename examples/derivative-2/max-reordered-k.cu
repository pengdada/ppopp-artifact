#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void curvi (double * __restrict__ in_r1, double *__restrict__ in_u1, double * __restrict__ in_u2, double *__restrict__ in_u3, double * __restrict__ in_mu, double * __restrict__ in_la, double * __restrict__ in_met1, double * __restrict__ in_met2, double * __restrict__ in_met3, double * __restrict__ in_met4, double * strx, double * stry, double c1, double c2, int N) {
	//Determing the block's indices
	int blockdim_k= (int)(blockDim.x);
	int k0 = (int)(blockIdx.x)*(blockdim_k);
	int k = max (k0, 0) + (int)(threadIdx.x);
	int blockdim_j= (int)(blockDim.y);
	int j0 = (int)(blockIdx.y)*(blockdim_j);
	int j = max (j0, 0) + (int)(threadIdx.y);

	double (*u1)[304][304] = (double (*)[304][304])in_u1;
	double (*u2)[304][304] = (double (*)[304][304])in_u2;
	double (*u3)[304][304] = (double (*)[304][304])in_u3;
	double (*mu)[304][304] = (double (*)[304][304])in_mu;
	double (*la)[304][304] = (double (*)[304][304])in_la;
	double (*r1)[304][304] = (double (*)[304][304])in_r1;
	double (*met1)[304][304] = (double (*)[304][304])in_met1;
	double (*met2)[304][304] = (double (*)[304][304])in_met2;
	double (*met3)[304][304] = (double (*)[304][304])in_met3;
	double (*met4)[304][304] = (double (*)[304][304])in_met4;

	if (j>=2 & k>=2 & j<=N-3 & k<=N-3) {
		for (int i=2; i<=N-3; i++) {
double _t_27_;
double _t_105_;
double _t_8_;
double _t_87_;
double _t_102_;
double _t_84_;
double _t_24_;
double _t_5_;
double _t_104_;
double _t_103_;
double _t_123_;
double _t_122_;
double _t_141_;
double _t_140_;
double _t_86_;
double _t_85_;
double _t_101_;
double _t_110_;
double _t_108_;
double _t_115_;
double _t_113_;
double _t_83_;
double _t_92_;
double _t_90_;
double _t_97_;
double _t_95_;
double _t_129_;
double _t_134_;
double _t_120_;
double _t_127_;
double _t_132_;
double _t_147_;
double _t_152_;
double _t_150_;
double _t_138_;
double _t_145_;
double _t_106_;
double _t_66_;
double _t_100_;
double _t_47_;
double _t_88_;
double _t_82_;
double _t_44_;
double _t_63_;
double _t_81_;
double _t_111_;
double _t_32_;
double _t_109_;
double _t_13_;
double _t_93_;
double _t_91_;
double _t_11_;
double _t_30_;
double _t_112_;
double _t_71_;
double _t_107_;
double _t_52_;
double _t_94_;
double _t_89_;
double _t_50_;
double _t_69_;
double _t_116_;
double _t_38_;
double _t_114_;
double _t_19_;
double _t_98_;
double _t_96_;
double _t_17_;
double _t_36_;
double _t_117_;
double _t_77_;
double _t_58_;
double _t_99_;
double _t_80_;
double _t_56_;
double _t_75_;
double _t_124_;
double _t_28_;
double _t_121_;
double _t_9_;
double _t_142_;
double _t_139_;
double _t_125_;
double _t_67_;
double _t_119_;
double _t_118_;
double _t_48_;
double _t_143_;
double _t_137_;
double _t_130_;
double _t_33_;
double _t_128_;
double _t_14_;
double _t_148_;
double _t_146_;
double _t_131_;
double _t_72_;
double _t_126_;
double _t_53_;
double _t_149_;
double _t_144_;
double _t_135_;
double _t_39_;
double _t_133_;
double _t_20_;
double _t_153_;
double _t_151_;
double _t_136_;
double _t_78_;
double _t_59_;
double _t_154_;
double _t_79_;
double r1ic0jc0kc0 = r1[i][j][k];
double _t_26_;
double _t_25_;
double _t_178_;
double _t_23_;
double _t_22_;
double _t_21_;
double _t_1_;
double _t_176_;
double _t_37_;
double _t_35_;
double _t_34_;
double _t_31_;
double _t_173_;
double _t_29_;
double _t_171_;
double _t_46_;
double _t_45_;
double _t_191_;
double _t_43_;
double _t_42_;
double _t_189_;
double _t_41_;
double _t_40_;
double _t_57_;
double _t_55_;
double _t_54_;
double _t_51_;
double _t_186_;
double _t_49_;
double _t_184_;
double _t_65_;
double _t_64_;
double _t_203_;
double _t_62_;
double _t_61_;
double _t_201_;
double _t_60_;
double _t_76_;
double _t_74_;
double _t_73_;
double _t_70_;
double _t_198_;
double _t_68_;
double _t_196_;
double _t_0_;
double _t_7_;
double _t_6_;
double _t_166_;
double _t_4_;
double _t_3_;
double _t_2_;
double _t_164_;
double _t_18_;
double _t_16_;
double _t_15_;
double _t_12_;
double _t_161_;
double _t_10_;
double _t_159_;
double _t_162_;
double _t_160_;
double _t_163_;
double _t_158_;
double _t_157_;
double _t_156_;
double _t_167_;
double _t_165_;
double _t_168_;
double _t_174_;
double _t_172_;
double _t_175_;
double _t_170_;
double _t_169_;
double _t_179_;
double _t_177_;
double _t_180_;
double _t_155_;
double _t_187_;
double _t_185_;
double _t_188_;
double _t_183_;
double _t_182_;
double _t_181_;
double _t_192_;
double _t_190_;
double _t_193_;
double _t_199_;
double _t_197_;
double _t_200_;
double _t_195_;
double _t_194_;
double _t_204_;
double _t_202_;
double _t_205_;

_t_27_ = -u1[i-2][j][k-2];
_t_105_ = -u1[i-2][j][k-2];
_t_105_ += u1[i-2][j][k+2];
_t_8_ = -u1[i-2][j][k+2];
_t_27_ += u1[i+2][j][k-2];
_t_87_ = -u1[i+2][j][k-2];
_t_8_ += u1[i+2][j][k+2];
_t_87_ += u1[i+2][j][k+2];
_t_102_ = c2 * _t_105_;
_t_84_ = c2 * _t_87_;
_t_24_ = c2 * _t_27_;
_t_5_ = c2 * _t_8_;
_t_104_ = 2.0 * mu[i-2][j][k];
_t_104_ += la[i-2][j][k];
_t_103_ = _t_104_ * met2[i-2][j][k];
_t_123_ = 2.0 * mu[i+1][j][k];
_t_123_ += la[i+1][j][k];
_t_122_ = _t_123_ * met2[i+1][j][k];
_t_141_ = 2.0 * mu[i-1][j][k];
_t_141_ += la[i-1][j][k];
_t_140_ = _t_141_ * met2[i-1][j][k];
_t_86_ = 2.0 * mu[i+2][j][k];
_t_86_ += la[i+2][j][k];
_t_85_ = _t_86_ * met2[i+2][j][k];
_t_101_ = _t_103_ * met1[i-2][j][k];
_t_110_ = la[i-2][j][k] * met3[i-2][j][k];
_t_108_ = _t_110_ * met1[i-2][j][k];
_t_115_ = la[i-2][j][k] * met4[i-2][j][k];
_t_113_ = _t_115_ * met1[i-2][j][k];
_t_83_ = _t_85_ * met1[i+2][j][k];
_t_92_ = la[i+2][j][k] * met3[i+2][j][k];
_t_90_ = _t_92_ * met1[i+2][j][k];
_t_97_ = la[i+2][j][k] * met4[i+2][j][k];
_t_95_ = _t_97_ * met1[i+2][j][k];
_t_129_ = la[i+1][j][k] * met3[i+1][j][k];
_t_134_ = la[i+1][j][k] * met4[i+1][j][k];
_t_120_ = _t_122_ * met1[i+1][j][k];
_t_127_ = _t_129_ * met1[i+1][j][k];
_t_132_ = _t_134_ * met1[i+1][j][k];
_t_147_ = la[i-1][j][k] * met3[i-1][j][k];
_t_152_ = la[i-1][j][k] * met4[i-1][j][k];
_t_150_ = _t_152_ * met1[i-1][j][k];
_t_138_ = _t_140_ * met1[i-1][j][k];
_t_145_ = _t_147_ * met1[i-1][j][k];
_t_106_ = -u1[i-2][j][k-1];
_t_66_ = -u1[i-2][j][k-1];
_t_106_ += u1[i-2][j][k+1];
_t_102_ += c1 * _t_106_;
_t_100_ = _t_101_ * _t_102_;
_t_47_ = -u1[i-2][j][k+1];
_t_66_ += u1[i+2][j][k-1];
_t_88_ = -u1[i+2][j][k-1];
_t_47_ += u1[i+2][j][k+1];
_t_88_ += u1[i+2][j][k+1];
_t_84_ += c1 * _t_88_;
_t_82_ = _t_83_ * _t_84_;
_t_44_ = c2 * _t_47_;
_t_63_ = c2 * _t_66_;
_t_81_ = _t_100_ * strx[i];
_t_81_ += _t_82_ * strx[i];
_t_111_ = -u2[i-2][j][k-2];
_t_32_ = -u2[i-2][j][k-2];
_t_111_ += u2[i-2][j][k+2];
_t_109_ = c2 * _t_111_;
_t_13_ = -u2[i-2][j][k+2];
_t_32_ += u2[i+2][j][k-2];
_t_93_ = -u2[i+2][j][k-2];
_t_13_ += u2[i+2][j][k+2];
_t_93_ += u2[i+2][j][k+2];
_t_91_ = c2 * _t_93_;
_t_11_ = c2 * _t_13_;
_t_30_ = c2 * _t_32_;
_t_112_ = -u2[i-2][j][k-1];
_t_71_ = -u2[i-2][j][k-1];
_t_112_ += u2[i-2][j][k+1];
_t_109_ += c1 * _t_112_;
_t_107_ = _t_108_ * _t_109_;
_t_81_ += _t_107_ * stry[j];
_t_52_ = -u2[i-2][j][k+1];
_t_71_ += u2[i+2][j][k-1];
_t_94_ = -u2[i+2][j][k-1];
_t_52_ += u2[i+2][j][k+1];
_t_94_ += u2[i+2][j][k+1];
_t_91_ += c1 * _t_94_;
_t_89_ = _t_90_ * _t_91_;
_t_81_ += _t_89_ * stry[j];
_t_50_ = c2 * _t_52_;
_t_69_ = c2 * _t_71_;
_t_116_ = -u3[i-2][j][k-2];
_t_38_ = -u3[i-2][j][k-2];
_t_116_ += u3[i-2][j][k+2];
_t_114_ = c2 * _t_116_;
_t_19_ = -u3[i-2][j][k+2];
_t_38_ += u3[i+2][j][k-2];
_t_98_ = -u3[i+2][j][k-2];
_t_19_ += u3[i+2][j][k+2];
_t_98_ += u3[i+2][j][k+2];
_t_96_ = c2 * _t_98_;
_t_17_ = c2 * _t_19_;
_t_36_ = c2 * _t_38_;
_t_117_ = -u3[i-2][j][k-1];
_t_77_ = -u3[i-2][j][k-1];
_t_117_ += u3[i-2][j][k+1];
_t_114_ += c1 * _t_117_;
_t_81_ += _t_113_ * _t_114_;
_t_58_ = -u3[i-2][j][k+1];
_t_77_ += u3[i+2][j][k-1];
_t_99_ = -u3[i+2][j][k-1];
_t_58_ += u3[i+2][j][k+1];
_t_99_ += u3[i+2][j][k+1];
_t_96_ += c1 * _t_99_;
_t_81_ += _t_95_ * _t_96_;
_t_80_ = c2 * _t_81_;
_t_56_ = c2 * _t_58_;
_t_75_ = c2 * _t_77_;
_t_124_ = -u1[i+1][j][k-2];
_t_28_ = u1[i+1][j][k-2];
_t_124_ += u1[i+1][j][k+2];
_t_121_ = c2 * _t_124_;
_t_9_ = u1[i+1][j][k+2];
_t_28_ -= u1[i-1][j][k-2];
_t_24_ += c1 * _t_28_;
_t_142_ = -u1[i-1][j][k-2];
_t_9_ -= u1[i-1][j][k+2];
_t_5_ += c1 * _t_9_;
_t_142_ += u1[i-1][j][k+2];
_t_139_ = c2 * _t_142_;
_t_125_ = -u1[i+1][j][k-1];
_t_67_ = u1[i+1][j][k-1];
_t_125_ += u1[i+1][j][k+1];
_t_121_ += c1 * _t_125_;
_t_119_ = _t_120_ * _t_121_;
_t_118_ = _t_119_ * strx[i];
_t_48_ = u1[i+1][j][k+1];
_t_67_ -= u1[i-1][j][k-1];
_t_63_ += c1 * _t_67_;
_t_143_ = -u1[i-1][j][k-1];
_t_48_ -= u1[i-1][j][k+1];
_t_44_ += c1 * _t_48_;
_t_143_ += u1[i-1][j][k+1];
_t_139_ += c1 * _t_143_;
_t_137_ = _t_138_ * _t_139_;
_t_118_ += _t_137_ * strx[i];
_t_130_ = -u2[i+1][j][k-2];
_t_33_ = u2[i+1][j][k-2];
_t_130_ += u2[i+1][j][k+2];
_t_128_ = c2 * _t_130_;
_t_14_ = u2[i+1][j][k+2];
_t_33_ -= u2[i-1][j][k-2];
_t_30_ += c1 * _t_33_;
_t_148_ = -u2[i-1][j][k-2];
_t_14_ -= u2[i-1][j][k+2];
_t_11_ += c1 * _t_14_;
_t_148_ += u2[i-1][j][k+2];
_t_146_ = c2 * _t_148_;
_t_131_ = -u2[i+1][j][k-1];
_t_72_ = u2[i+1][j][k-1];
_t_131_ += u2[i+1][j][k+1];
_t_128_ += c1 * _t_131_;
_t_126_ = _t_127_ * _t_128_;
_t_118_ += _t_126_ * stry[j];
_t_53_ = u2[i+1][j][k+1];
_t_72_ -= u2[i-1][j][k-1];
_t_69_ += c1 * _t_72_;
_t_149_ = -u2[i-1][j][k-1];
_t_53_ -= u2[i-1][j][k+1];
_t_50_ += c1 * _t_53_;
_t_149_ += u2[i-1][j][k+1];
_t_146_ += c1 * _t_149_;
_t_144_ = _t_145_ * _t_146_;
_t_118_ += _t_144_ * stry[j];
_t_135_ = -u3[i+1][j][k-2];
_t_39_ = u3[i+1][j][k-2];
_t_135_ += u3[i+1][j][k+2];
_t_133_ = c2 * _t_135_;
_t_20_ = u3[i+1][j][k+2];
_t_39_ -= u3[i-1][j][k-2];
_t_36_ += c1 * _t_39_;
_t_153_ = -u3[i-1][j][k-2];
_t_20_ -= u3[i-1][j][k+2];
_t_17_ += c1 * _t_20_;
_t_153_ += u3[i-1][j][k+2];
_t_151_ = c2 * _t_153_;
_t_136_ = -u3[i+1][j][k-1];
_t_78_ = u3[i+1][j][k-1];
_t_136_ += u3[i+1][j][k+1];
_t_133_ += c1 * _t_136_;
_t_118_ += _t_132_ * _t_133_;
_t_59_ = u3[i+1][j][k+1];
_t_78_ -= u3[i-1][j][k-1];
_t_75_ += c1 * _t_78_;
_t_154_ = -u3[i-1][j][k-1];
_t_59_ -= u3[i-1][j][k+1];
_t_56_ += c1 * _t_59_;
_t_154_ += u3[i-1][j][k+1];
_t_151_ += c1 * _t_154_;
_t_118_ += _t_150_ * _t_151_;
_t_80_ += c1 * _t_118_;
_t_79_ = _t_80_ * stry[j];
r1ic0jc0kc0 += _t_79_;
_t_26_ = 2.0 * mu[i][j][k-2];
_t_26_ += la[i][j][k-2];
_t_25_ = _t_26_ * met2[i][j][k-2];
_t_178_ = la[i][j][k-2] * met2[i][j][k-2];
_t_23_ = _t_25_ * met1[i][j][k-2];
_t_22_ = _t_23_ * _t_24_;
_t_21_ = _t_22_ * strx[i];
_t_1_ = _t_21_ * stry[j];
_t_176_ = _t_178_ * met1[i][j][k-2];
_t_37_ = mu[i][j][k-2] * met4[i][j][k-2];
_t_35_ = _t_37_ * met1[i][j][k-2];
_t_34_ = _t_35_ * _t_36_;
_t_1_ += _t_34_ * stry[j];
_t_31_ = mu[i][j][k-2] * met3[i][j][k-2];
_t_173_ = mu[i][j][k-2] * met3[i][j][k-2];
_t_29_ = _t_31_ * met1[i][j][k-2];
_t_1_ += _t_29_ * _t_30_;
_t_171_ = _t_173_ * met1[i][j][k-2];
_t_46_ = 2.0 * mu[i][j][k+1];
_t_46_ += la[i][j][k+1];
_t_45_ = _t_46_ * met2[i][j][k+1];
_t_191_ = la[i][j][k+1] * met2[i][j][k+1];
_t_43_ = _t_45_ * met1[i][j][k+1];
_t_42_ = _t_43_ * _t_44_;
_t_189_ = _t_191_ * met1[i][j][k+1];
_t_41_ = _t_42_ * strx[i+2];
_t_40_ = _t_41_ * stry[j];
_t_57_ = mu[i][j][k+1] * met4[i][j][k+1];
_t_55_ = _t_57_ * met1[i][j][k+1];
_t_54_ = _t_55_ * _t_56_;
_t_40_ += _t_54_ * stry[j];
_t_51_ = mu[i][j][k+1] * met3[i][j][k+1];
_t_186_ = mu[i][j][k+1] * met3[i][j][k+1];
_t_49_ = _t_51_ * met1[i][j][k+1];
_t_40_ += _t_49_ * _t_50_;
_t_184_ = _t_186_ * met1[i][j][k+1];
_t_65_ = 2.0 * mu[i][j][k-1];
_t_65_ += la[i][j][k-1];
_t_64_ = _t_65_ * met2[i][j][k-1];
_t_203_ = la[i][j][k-1] * met2[i][j][k-1];
_t_62_ = _t_64_ * met1[i][j][k-1];
_t_61_ = _t_62_ * _t_63_;
_t_201_ = _t_203_ * met1[i][j][k-1];
_t_60_ = _t_61_ * strx[i-2];
_t_40_ += _t_60_ * stry[j];
_t_76_ = mu[i][j][k-1] * met4[i][j][k-1];
_t_74_ = _t_76_ * met1[i][j][k-1];
_t_73_ = _t_74_ * _t_75_;
_t_40_ += _t_73_ * stry[j];
_t_70_ = mu[i][j][k-1] * met3[i][j][k-1];
_t_198_ = mu[i][j][k-1] * met3[i][j][k-1];
_t_68_ = _t_70_ * met1[i][j][k-1];
_t_40_ += _t_68_ * _t_69_;
_t_196_ = _t_198_ * met1[i][j][k-1];
_t_0_ = c1 * _t_40_;
_t_7_ = 2.0 * mu[i][j][k+2];
_t_7_ += la[i][j][k+2];
_t_6_ = _t_7_ * met2[i][j][k+2];
_t_166_ = la[i][j][k+2] * met2[i][j][k+2];
_t_4_ = _t_6_ * met1[i][j][k+2];
_t_3_ = _t_4_ * _t_5_;
_t_2_ = _t_3_ * strx[i];
_t_1_ += _t_2_ * stry[j];
_t_164_ = _t_166_ * met1[i][j][k+2];
_t_18_ = mu[i][j][k+2] * met4[i][j][k+2];
_t_16_ = _t_18_ * met1[i][j][k+2];
_t_15_ = _t_16_ * _t_17_;
_t_1_ += _t_15_ * stry[j];
_t_12_ = mu[i][j][k+2] * met3[i][j][k+2];
_t_161_ = mu[i][j][k+2] * met3[i][j][k+2];
_t_10_ = _t_12_ * met1[i][j][k+2];
_t_1_ += _t_10_ * _t_11_;
_t_0_ += c2 * _t_1_;
r1ic0jc0kc0 += _t_0_;
_t_159_ = _t_161_ * met1[i][j][k+2];
_t_162_ = -u1[i][j-2][k+2];
_t_162_ += u1[i][j+2][k+2];
_t_160_ = c2 * _t_162_;
_t_163_ = -u1[i][j-1][k+2];
_t_163_ += u1[i][j+1][k+2];
_t_160_ += c1 * _t_163_;
_t_158_ = _t_159_ * _t_160_;
_t_157_ = _t_158_ * stry[j+2];
_t_156_ = _t_157_ * strx[i];
_t_167_ = -u2[i][j-2][k+2];
_t_167_ += u2[i][j+2][k+2];
_t_165_ = c2 * _t_167_;
_t_168_ = -u2[i][j-1][k+2];
_t_168_ += u2[i][j+1][k+2];
_t_165_ += c1 * _t_168_;
_t_156_ += _t_164_ * _t_165_;
_t_174_ = -u1[i][j-2][k-2];
_t_174_ += u1[i][j+2][k-2];
_t_172_ = c2 * _t_174_;
_t_175_ = -u1[i][j-1][k-2];
_t_175_ += u1[i][j+1][k-2];
_t_172_ += c1 * _t_175_;
_t_170_ = _t_171_ * _t_172_;
_t_169_ = _t_170_ * stry[j];
_t_156_ += _t_169_ * strx[i];
_t_179_ = -u2[i][j-2][k-2];
_t_179_ += u2[i][j+2][k-2];
_t_177_ = c2 * _t_179_;
_t_180_ = -u2[i][j-1][k-2];
_t_180_ += u2[i][j+1][k-2];
_t_177_ += c1 * _t_180_;
_t_156_ += _t_176_ * _t_177_;
_t_155_ = c2 * _t_156_;
_t_187_ = -u1[i][j-2][k+1];
_t_187_ += u1[i][j+2][k+1];
_t_185_ = c2 * _t_187_;
_t_188_ = -u1[i][j-1][k+1];
_t_188_ += u1[i][j+1][k+1];
_t_185_ += c1 * _t_188_;
_t_183_ = _t_184_ * _t_185_;
_t_182_ = _t_183_ * stry[j-2];
_t_181_ = _t_182_ * strx[i];
_t_192_ = -u2[i][j-2][k+1];
_t_192_ += u2[i][j+2][k+1];
_t_190_ = c2 * _t_192_;
_t_193_ = -u2[i][j-1][k+1];
_t_193_ += u2[i][j+1][k+1];
_t_190_ += c1 * _t_193_;
_t_181_ += _t_189_ * _t_190_;
_t_199_ = -u1[i][j-2][k-1];
_t_199_ += u1[i][j+2][k-1];
_t_197_ = c2 * _t_199_;
_t_200_ = -u1[i][j-1][k-1];
_t_200_ += u1[i][j+1][k-1];
_t_197_ += c1 * _t_200_;
_t_195_ = _t_196_ * _t_197_;
_t_194_ = _t_195_ * stry[j];
_t_181_ += _t_194_ * strx[i];
_t_204_ = -u2[i][j-2][k-1];
_t_204_ += u2[i][j+2][k-1];
_t_202_ = c2 * _t_204_;
_t_205_ = -u2[i][j-1][k-1];
_t_205_ += u2[i][j+1][k-1];
_t_202_ += c1 * _t_205_;
_t_181_ += _t_201_ * _t_202_;
_t_155_ += c1 * _t_181_;
r1ic0jc0kc0 += _t_155_;
r1[i][j][k] = r1ic0jc0kc0;

			r1[i][j][k] += c2*(
					mu[i][j+2][k]*met3[i][j+2][k]*met1[i][j+2][k]*(
						c2*(u1[i][j+2][k+2]-u1[i][j+2][k-2]) +
						c1*(u1[i][j+2][k+1]-u1[i][j+2][k-1])   )*stry[j+1]*strx[i]
					+ mu[i][j+2][k]*met2[i][j+2][k]*met1[i][j+2][k]*(
						c2*(u2[i][j+2][k+2]-u2[i][j+2][k-2]) +
						c1*(u2[i][j+2][k+1]-u2[i][j+2][k-1])  )
					+ ( mu[i][j-2][k]*met3[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u1[i][j-2][k+2]-u1[i][j-2][k-2]) +
							c1*(u1[i][j-2][k+1]-u1[i][j-2][k-1])  )*stry[j]*strx[i]
						+ mu[i][j-2][k]*met2[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u2[i][j-2][k+2]-u2[i][j-2][k-2]) +
							c1*(u2[i][j-2][k+1]-u2[i][j-2][k-1])   ) )
					) + c1*(
						mu[i][j+1][k]*met3[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u1[i][j+1][k+2]-u1[i][j+1][k-2]) +
							c1*(u1[i][j+1][k+1]-u1[i][j+1][k-1]) )*stry[j-1]*strx[i]
						+ mu[i][j+1][k]*met2[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u2[i][j+1][k+2]-u2[i][j+1][k-2]) +
							c1*(u2[i][j+1][k+1]-u2[i][j+1][k-1]) )
						+ ( mu[i][j-1][k]*met3[i][j-1][k]*met1[i][j-1][k]*(
								c2*(u1[i][j-1][k+2]-u1[i][j-1][k-2]) +
								c1*(u1[i][j-1][k+1]-u1[i][j-1][k-1]) )*stry[j]*strx[i]
							+ mu[i][j-1][k]*met2[i][j-1][k]*met1[i][j-1][k]*(
								c2*(u2[i][j-1][k+2]-u2[i][j-1][k-2]) +
								c1*(u2[i][j-1][k+1]-u2[i][j-1][k-1]) ) ) );

			r1[i][j][k] +=
				c2*(  mu[i][j+2][k]*met1[i][j+2][k]*met1[i][j+2][k]*(
							c2*(u2[i+2][j+2][k]-u2[i-2][j+2][k]) +
							c1*(u2[i+1][j+2][k]-u2[i-1][j+2][k])    )
						+  mu[i][j-2][k]*met1[i][j-2][k]*met1[i][j-2][k]*(
							c2*(u2[i+2][j-2][k]-u2[i-2][j-2][k])+
							c1*(u2[i+1][j-2][k]-u2[i-1][j-2][k])     )
				   ) +
				c1*(  mu[i][j+1][k]*met1[i][j+1][k]*met1[i][j+1][k]*(
							c2*(u2[i+2][j+1][k]-u2[i-2][j+1][k]) +
							c1*(u2[i+1][j+1][k]-u2[i-1][j+1][k])  )
						+ mu[i][j-1][k]*met1[i][j-1][k]*met1[i][j-1][k]*(
							c2*(u2[i+2][j-1][k]-u2[i-2][j-1][k]) +
							c1*(u2[i+1][j-1][k]-u2[i-1][j-1][k])))
				+
				c2*(  la[i+2][j][k]*met1[i+2][j][k]*met1[i+2][j][k]*(
							c2*(u2[i+2][j+2][k]-u2[i+2][j-2][k]) +
							c1*(u2[i+2][j+1][k]-u2[i+2][j-1][k])    )
						+ la[i-2][j][k]*met1[i-2][j][k]*met1[i-2][j][k]*(
							c2*(u2[i-2][j+2][k]-u2[i-2][j-2][k])+
							c1*(u2[i-2][j+1][k]-u2[i-2][j-1][k])     )
				   ) +
				c1*(  la[i+1][j][k]*met1[i+1][j][k]*met1[i+1][j][k]*(
							c2*(u2[i+1][j+2][k]-u2[i+1][j-2][k]) +
							c1*(u2[i+1][j+1][k]-u2[i+1][j-1][k])  )
						+ la[i-1][j][k]*met1[i-1][j][k]*met1[i-1][j][k]*(
							c2*(u2[i-1][j+2][k]-u2[i-1][j-2][k]) +
							c1*(u2[i-1][j+1][k]-u2[i-1][j-1][k])));

		} 
	}
}

extern "C" void host_code (double *h_r1, double *h_u1, double *h_u2, double *h_u3,  double *h_mu, double *h_la, double *h_met1, double *h_met2, double *h_met3, double *h_met4, double *h_strx, double *h_stry, double c1, double c2, int N) {
	double *r1;
	hipMalloc (&r1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for r1\n");
	hipMemcpy (r1, h_r1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u1;
	hipMalloc (&u1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u1\n");
	hipMemcpy (u1, h_u1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u2;
	hipMalloc (&u2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u2\n");
	hipMemcpy (u2, h_u2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *u3;
	hipMalloc (&u3, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for u3\n");
	hipMemcpy (u3, h_u3, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *mu;
	hipMalloc (&mu, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for mu\n");
	hipMemcpy (mu, h_mu, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *la;
	hipMalloc (&la, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for la\n");
	hipMemcpy (la, h_la, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met1;
	hipMalloc (&met1, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met1\n");
	hipMemcpy (met1, h_met1, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met2;
	hipMalloc (&met2, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met2\n");
	hipMemcpy (met2, h_met2, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met3;
	hipMalloc (&met3, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met3\n");
	hipMemcpy (met3, h_met3, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *met4;
	hipMalloc (&met4, sizeof(double)*N*N*N);
	check_error ("Failed to allocate device memory for met4\n");
	hipMemcpy (met4, h_met4, sizeof(double)*N*N*N, hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double)*N);
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double)*N, hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double)*N);
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 blockconfig (16, 8);
	dim3 gridconfig (ceil(N, blockconfig.x), ceil(N, blockconfig.y), 1);

	curvi <<<gridconfig, blockconfig>>> (r1, u1, u2, u3, mu, la, met1, met2, met3, met4, strx, stry, c1, c2, N);
	hipMemcpy (h_r1, r1, sizeof(double)*N*N*N, hipMemcpyDeviceToHost);
}
